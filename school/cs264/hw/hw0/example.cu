#include "hip/hip_runtime.h"
/* 
   
   Example CUDA code for Problem Set 0, CS 264, Harvard, Fall 2009.  

	 Takes an input string and mangles it using the current date and
	 time on the CPU and on the GPU.  Demonstrates device initialization
	 and error checking with libcutil, host<=>device memory transfers,
	 and CUDA kernel invocation.

   To compile: 
	     
        nvcc example.cu -o example -I$CUDASDK_HOME/common/inc	\
            -L$CUDASDK_HOME/lib/linux -lcutil

	 Usage: 
  
        example -string="<str>" {-device=<dev>} ,
	  
	 where <str> is the input string, and, optionally, <dev> is the device 
	 number.

	 Kevin Dale <dale@eecs.harvard.edu>
	 08.20.09
*/

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "cutil.h"

/* forward declarations */
__global__ void mangleGPU(char*,char*,int,int); // GPU kernel prototype
           void mangleCPU(char*,char*,int,int); // CPU prototype

/* macro to mangle an n-length char string, shared between CPU and GPU code */
#define MANGLE(instr,outstr,i,n,x) \
	((outstr)[(i)]=(((instr)[((i)+(x)+(instr)[(i)])%(n)])))

/* main driver */
int main(int argc, char** argv){

	// - initialize device
	CUT_DEVICE_INIT(argc,argv);

	// - read command-line args
	char *str;
	cutGetCmdLineArgumentstr(argc,(const char**)argv,"string",&str);
	int n=strlen(str);

	// - get the current time
	time_t now=time(0);
	char *nowstring=asctime(localtime(&now));

	// - allocate memory on the device
	char *d_str_in, *d_str_out;
	hipMalloc((void**)&d_str_in, n*sizeof(char));
	hipMalloc((void**)&d_str_out,n*sizeof(char));
	CUT_CHECK_ERROR("initial alloc");

	// - copy data to device
	hipMemcpy(d_str_in,str,n*sizeof(char),hipMemcpyHostToDevice);
	CUT_CHECK_ERROR("copy host to device");

	// - invoke the kernel
	int nblocks=1, nthreads=n;
	mangleGPU<<<nblocks,nthreads>>>(d_str_in,d_str_out,n,(int)now);
	CUT_CHECK_ERROR("kernel invocation");

	// - copy from device to main memory
	char *gpu_result=(char*)malloc((n+1)*sizeof(char));
	hipMemcpy(gpu_result,d_str_out,n*sizeof(char),hipMemcpyDeviceToHost); 
	CUT_CHECK_ERROR("copy device to host");

	// - invoke the equivalent CPU function
	char *cpu_result=(char*)malloc((n+1)*sizeof(char));	
	mangleCPU(str,cpu_result,n,(int)now);

	// - put null terminating character at end of each result
	gpu_result[n]=char(0);
	cpu_result[n]=char(0);

	// - report results
	printf("Current date/time: (%d) %s",now,nowstring);
	printf("Input string:      %s\n",str,n);
	printf("CPU result:        %s\n",cpu_result);
	printf("GPU result:        %s\n",gpu_result);

	// - cleanup and return
	hipFree(d_str_in);
	hipFree(d_str_out);
	free(gpu_result);
	free(cpu_result);

	return 0;
}

/* CUDA device kernel */
__global__ void mangleGPU(char* instr, char *outstr, int len, int x){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	MANGLE(instr,outstr,i,len,x);
}

/* CPU implementation */
void mangleCPU(char *instr, char *outstr, int len, int x){
	for(int i=0; i<len; i++)
		MANGLE(instr,outstr,i,len,x);
}
