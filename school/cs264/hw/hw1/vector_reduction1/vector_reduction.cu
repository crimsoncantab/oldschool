#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Adapted by Nady Obeid, Xiao-Long Wu, and I-Jui Sung, UIUC */

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include <vector_reduction_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

int ReadFile(float*, char* file_name, int num_elements);
float computeOnDevice(float* h_data, int array_mem_size);

extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

void randomize(float * h_data, int num_elements) {
    // initialize the input data on the host to be integer values
    // between 0 and 1000
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        srand(time(NULL));
        h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Run naive scan test
////////////////////////////////////////////////////////////////////////////////
void runTest( int argc, char** argv) 
{
    int num_elements = NUM_ELEMENTS;
    unsigned int array_mem_size;
    float* h_data;
    int errorM = 0;
    int num;


    // * No arguments: Randomly generate input data and compare against the 
    //   host's result, using default NUM_ELEMENTS
    // * One argument: Randomly generate arg1 input elements
    // * Two arguments: first is num elements, second is file containing vals
    switch(argc-1)
    {      
        case 2: //Two arguments
            num = atoi(argv[1]);
            if (num != 0) {
                num_elements = num;
                array_mem_size = sizeof( float) * num_elements;
                h_data = (float*) malloc( array_mem_size);           
                errorM = ReadFile(h_data, argv[2], num_elements);
                if(errorM != 1)
                {
                    printf("Error reading input file!\n");
                    exit(1);
                }
            }
            else {
                printf("first arg must be a non-zero number");
                exit(-1);
            }
        
        case 1:  // One Argument
            num = atoi(argv[1]);
            
            if (num != 0) {
                num_elements = num;
            }
            
            array_mem_size = sizeof( float) * num_elements;
            h_data = (float*) malloc( array_mem_size);
            randomize(h_data, num_elements);
            
            break;  
        default:  // No Arguments or one argument
        
            array_mem_size = sizeof( float) * num_elements;
            h_data = (float*) malloc( array_mem_size);
            randomize(h_data, num_elements);
            
            break;  
    }
    // compute reference solution
    float reference = 0.0f;  
    computeGold(&reference , h_data, num_elements);

    // **===-------- Modify the body of this function -----------===**
    float result = computeOnDevice(h_data, num_elements);
    // **===-----------------------------------------------------------===**


    // We can use an epsilon of 0 since values are integral and in a range 
    // that can be exactly represented
    float epsilon = 0.0f;
    unsigned int result_regtest = (abs(result - reference) <= epsilon);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    printf( "device: %f  host: %f\n", result, reference);
    // cleanup memory
    free( h_data);
}


int ReadFile(float* M, char* file_name, int num_elements)
{
    unsigned int elements_read = num_elements;
    if (cutReadFilef(file_name, &M, &elements_read, true))
        return 1;
    else
        return 0;
}

// **===----------------- Modify this function ---------------------===**
// Take h_data from host, copy it to device, setup grid and thread 
// dimensions, execute kernel , and copy result back to h_data.
// Note: float* h_data is both the input and the output of this function.
float computeOnDevice(float* h_data, int n)
{
    float result = 0.0;

    // =====================================================
    // Code segment 1:

    // * Allocate CUDA device memory
    float * d_data;
    hipMalloc(&d_data, n * sizeof(float));
	CUT_CHECK_ERROR("initial alloc");

    // * Copy input data from host memory to CUDA device memory
    hipMemcpy(d_data, h_data, n * sizeof(float), hipMemcpyHostToDevice);
	CUT_CHECK_ERROR("copy host to device");

    int offset = 1;
    int n_temp = n;
    
    while (n_temp != 1 /*last value is the answer */) {
        //divide-round-up for last block holding remainder
        int gridD = (n_temp + BLOCK_SIZE - 1) / BLOCK_SIZE;
        
        // * Setup block and grid sizes
        dim3 blockDim(BLOCK_SIZE);
        dim3 gridDim(gridD);

        printf("%d, %d, %d\n", gridDim.x, n_temp, offset);
        // * Invoke the kernel
        reduction<<<gridDim, blockDim>>>(d_data, n_temp, offset);
        CUT_CHECK_ERROR("kernel run\n");

        hipDeviceSynchronize();
        hipMemcpy(&result, d_data, sizeof(float), hipMemcpyDeviceToHost);
        printf("result %f\n",result);
        //we have one value per block now
        n_temp = gridD;
        //outputs of last run in progressively more distant locations
        offset *= BLOCK_SIZE;
        
    }

    // * Copy results from CUDA device memory back to host memory
    hipMemcpy(&result, d_data, sizeof(float), hipMemcpyDeviceToHost);

    // * Free device memory
    hipFree(d_data);


    return result;
}

