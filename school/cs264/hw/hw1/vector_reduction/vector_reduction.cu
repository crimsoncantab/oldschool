#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Adapted by Nady Obeid, Xiao-Long Wu, and I-Jui Sung, UIUC */

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include <vector_reduction_kernel.cu>

// For simplicity, just to get the idea in this MP, we're fixing the problem size to 512 elements.
#define NUM_ELEMENTS 512

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

int ReadFile(float*, char* file_name);
float computeOnDevice(float* h_data, int array_mem_size);

extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//! Run naive scan test
////////////////////////////////////////////////////////////////////////////////
void runTest( int argc, char** argv) 
{
    int num_elements = NUM_ELEMENTS;
    int errorM = 0;

    const unsigned int array_mem_size = sizeof( float) * num_elements;

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( array_mem_size);

    // * No arguments: Randomly generate input data and compare against the 
    //   host's result.
    // * One argument: Read the input data array from the given file.
    switch(argc-1)
    {      
        case 1:  // One Argument
            errorM = ReadFile(h_data, argv[1]);
            if(errorM != 1)
            {
                printf("Error reading input file!\n");
                exit(1);
            }
            break;

        default:  // No Arguments or one argument
            // initialize the input data on the host to be integer values
            // between 0 and 1000
            for( unsigned int i = 0; i < num_elements; ++i) 
            {
                srand(time(NULL));
                h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
            }
            break;  
    }
    // compute reference solution
    float reference = 0.0f;  
    computeGold(&reference , h_data, num_elements);

    // **===-------- Modify the body of this function -----------===**
    float result = computeOnDevice(h_data, num_elements);
    // **===-----------------------------------------------------------===**


    // We can use an epsilon of 0 since values are integral and in a range 
    // that can be exactly represented
    float epsilon = 0.0f;
    unsigned int result_regtest = (abs(result - reference) <= epsilon);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    printf( "device: %f  host: %f\n", result, reference);
    // cleanup memory
    free( h_data);
}


int ReadFile(float* M, char* file_name)
{
    unsigned int elements_read = NUM_ELEMENTS;
    if (cutReadFilef(file_name, &M, &elements_read, true))
        return 1;
    else
        return 0;
}

// **===----------------- Modify this function ---------------------===**
// Take h_data from host, copy it to device, setup grid and thread 
// dimensions, execute kernel , and copy result back to h_data.
// Note: float* h_data is both the input and the output of this function.
float computeOnDevice(float* h_data, int n)
{
    float result = 0.0;

    // =====================================================
    // Code segment 1:

    // * Allocate CUDA device memory
    float * d_data;
    hipMalloc(&d_data, n * sizeof(float));
	CUT_CHECK_ERROR("initial alloc");

    // * Copy input data from host memory to CUDA device memory
    hipMemcpy(d_data, h_data, n * sizeof(float), hipMemcpyHostToDevice);
	CUT_CHECK_ERROR("copy host to device");

    // * Setup block and grid sizes
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim(1);

    // * Invoke the kernel
    reduction<<<gridDim, blockDim>>>(d_data, n);
    CUT_CHECK_ERROR("kernel run\n");
    hipDeviceSynchronize();

    // * Copy results from CUDA device memory back to host memory
    hipMemcpy(&result, d_data, sizeof(float), hipMemcpyDeviceToHost);

    // * Free device memory
    hipFree(d_data);


    return result;
}

